#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "../CudaControl/Helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "SwapGates.hpp"

int swap(custatevecHandle_t &handle,
         const int nIndexBits,
         const int2 bitSwaps[],
         const int nBitSwaps,
         hipDoubleComplex *d_sv)

{

    // swap the state vector elements only if 1st qubit is 1
    const int maskLen = 0;
    int maskBitString[] = {};
    int maskOrdering[] = {};
    // bit swap
    CHECK_CUSTATEVECTOR(custatevecSwapIndexBits(
        handle, d_sv, HIP_C_64F, nIndexBits, bitSwaps, nBitSwaps,
        maskBitString, maskOrdering, maskLen));

    return hipSuccess;
}