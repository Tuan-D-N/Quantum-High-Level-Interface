#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "ApplyGates.hpp"

int applyGatesGeneral(custatevecHandle_t &handle,
                      const int nIndexBits,
                      const hipDoubleComplex matrix[],
                      const int adjoint,
                      const int targets[],
                      const int nTargets,
                      const int controls[],
                      const int nControls,
                      hipDoubleComplex *d_sv,
                      void *extraWorkspace,
                      size_t extraWorkspaceSizeInBytes)
{

    // check the size of external workspace
    HANDLE_ERROR(custatevecApplyMatrixGetWorkspaceSize(
        handle, HIP_C_64F, nIndexBits, matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
        adjoint, nTargets, nControls, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes));

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
    {
        std::cout << "Extra space needed: " << extraWorkspaceSizeInBytes << " Bytes";
        if (extraWorkspace != nullptr)
        {
            HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
        }
        HANDLE_CUDA_ERROR(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));
    }
    // apply gate
    HANDLE_ERROR(custatevecApplyMatrix(
        handle, d_sv, HIP_C_64F, nIndexBits, matrix, HIP_C_64F,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls, nullptr,
        nControls, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
    if (extraWorkspace != nullptr)
    {
        HANDLE_CUDA_ERROR(hipFree(extraWorkspace));
    }
    return hipSuccess;
}