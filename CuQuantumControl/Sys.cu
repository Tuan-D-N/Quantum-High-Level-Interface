#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "../CudaControl/Helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "ApplyGates.hpp"
#include "ApplySampler.hpp"
#include "QftStateVec.hpp"
#include "../functionality/fftShift.hpp"
#include "../functionality/ClockTimer.hpp"
#include <cstring>

int runner()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;

    hipDoubleComplex h_sv[] = {{1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}, {8, 0}};
    hipDoubleComplex *d_sv;
    CHECK_CUDA(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    // Initialize the values
    std::memcpy(d_sv, &h_sv, nSvSize * sizeof(hipDoubleComplex));

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    fftshift1D(d_sv, nSvSize);
    CHECK_CUDA(static_cast<hipError_t>(ApplyQFTOnStateVector(d_sv, nIndexBits)));
    fftshift1D(d_sv, nSvSize);

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    CHECK_CUDA(hipFree(d_sv));

    return hipSuccess;
}
int runner2()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;

    hipDoubleComplex h_sv[] = {{1, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}};
    hipDoubleComplex *d_sv;
    CHECK_CUDA(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    // Initialize the values
    std::memcpy(d_sv, &h_sv, nSvSize * sizeof(hipDoubleComplex));

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    custatevecHandle_t handle;
    CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
    void *extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    applyX<3>(handle, nIndexBits, adjoint, {0, 1, 2}, d_sv, extraWorkspace, extraWorkspaceSizeInBytes);
    // hipDoubleComplex matrix[] = XMat;
    // const int target[] = {0};
    // const int control[] = {};
    // applyGatesGeneral(handle, nIndexBits, matrix, adjoint, target, 1, control, 0, d_sv, extraWorkspace, extraWorkspaceSizeInBytes);
    // applyX(handle, nIndexBits, adjoint, target[0], control, sizeof(control) / sizeof(control[0]), d_sv, extraWorkspace, extraWorkspaceSizeInBytes);

    CHECK_CUSTATEVECTOR(custatevecDestroy(handle));

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";
    CHECK_CUDA(hipFree(d_sv));

    return hipSuccess;
}

template <int N>
struct rangeArray
{
    constexpr rangeArray() : arr()
    {
        for (auto i = 0; i != N; ++i)
            arr[i] = i;
    }
    std::array<int, N> arr;
};

int runner3()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;
    {
        Timer("Grover Cuquantum C++ qubits = " + std::to_string(nIndexBits));

        // Make the statevector -------------------------------------------------------------------------------
        hipDoubleComplex *d_sv;
        CHECK_CUDA(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));
        d_sv[0] = {1, 0};
        for (int i = 1; i < nSvSize; ++i)
        {
            d_sv[i] = {0, 0};
        }
        // Make the statevector -------------------------------------------------------------------------------

        // Grover ----------------------------------------------------------------------------------------
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Algo ------------------------------------------------------------
        constexpr auto allQubit = rangeArray<nIndexBits>().arr;
        constexpr auto allQubitExceptLast = rangeArray<nIndexBits - 1>().arr;

        CHECK_BROAD_ERROR(applyH<nIndexBits>(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

        for (int i = 0; i < 10; ++i)
        {
            // Mark
            constexpr int markTarget = nIndexBits - 1; // lastQubit
            CHECK_BROAD_ERROR(applyZ<nIndexBits - 1>(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

            // Diffusion
            CHECK_BROAD_ERROR(applyH<nIndexBits>(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyX<nIndexBits>(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyZ<nIndexBits - 1>(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyX<nIndexBits>(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyH<nIndexBits>(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
        }

        // Algo ------------------------------------------------------------
        CHECK_BROAD_ERROR(custatevecDestroy(handle));
        if (extraWorkspace != nullptr)
            CHECK_CUDA(hipFree(extraWorkspace));

        // Grover ----------------------------------------------------------------------------------------
        CHECK_CUDA(hipFree(d_sv));
    }

    return hipSuccess;
}