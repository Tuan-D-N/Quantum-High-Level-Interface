#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "ApplyGates.hpp"
#include "QftStateVec.hpp"
#include "../functionality/fftShift.hpp"
#include <cstring>
int runner()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;

    hipDoubleComplex h_sv[] = {{1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}, {8, 0}};
    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    // Initialize the values
    std::memcpy(d_sv, &h_sv, nSvSize * sizeof(hipDoubleComplex));

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    fftshift1D(d_sv, nSvSize);
    HANDLE_CUDA_ERROR(static_cast<hipError_t>(ApplyQFTOnStateVector(d_sv, nIndexBits)));
    fftshift1D(d_sv, nSvSize);

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    HANDLE_CUDA_ERROR(hipFree(d_sv));

    return hipSuccess;
}
int runner2()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;

    hipDoubleComplex h_sv[] = {{1, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}, {0, 0}};
    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    // Initialize the values
    std::memcpy(d_sv, &h_sv, nSvSize * sizeof(hipDoubleComplex));

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    custatevecHandle_t handle;
    HANDLE_ERROR(custatevecCreate(&handle));
    void *extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    applyX<3>(handle, nIndexBits, adjoint, {0, 1, 2}, d_sv, extraWorkspace, extraWorkspaceSizeInBytes);
    // hipDoubleComplex matrix[] = XMat;
    // const int target[] = {0};
    // const int control[] = {};
    // applyGatesGeneral(handle, nIndexBits, matrix, adjoint, target, 1, control, 0, d_sv, extraWorkspace, extraWorkspaceSizeInBytes);
    // applyX(handle, nIndexBits, adjoint, target[0], control, sizeof(control) / sizeof(control[0]), d_sv, extraWorkspace, extraWorkspaceSizeInBytes);

    HANDLE_ERROR(custatevecDestroy(handle));

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";
    HANDLE_CUDA_ERROR(hipFree(d_sv));

    return hipSuccess;
}