/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "ApplyGates.hpp"

int runner()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int nTargets = 1;
    const int nControls = 2;
    const int adjoint = 0;

    const int targets[] = {2};
    const int controls[] = {0, 1};

    hipDoubleComplex matrix[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    // Initialize the values
    d_sv[0] = make_hipDoubleComplex(1.0, 0.0);
    d_sv[1] = make_hipDoubleComplex(2.0, 0.0);
    d_sv[2] = make_hipDoubleComplex(3.0, 0.0);
    d_sv[3] = make_hipDoubleComplex(4.0, 0.0);
    d_sv[4] = make_hipDoubleComplex(5.0, 0.0);
    d_sv[5] = make_hipDoubleComplex(6.0, 0.0);
    d_sv[6] = make_hipDoubleComplex(7.0, 0.0);
    d_sv[7] = make_hipDoubleComplex(8.0, 0.0);

    custatevecHandle_t handle;
    HANDLE_ERROR(custatevecCreate(&handle));
    void *extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    applyX(handle, nIndexBits, (int)false, 0, d_sv, extraWorkspace, extraWorkspaceSizeInBytes);

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }

    HANDLE_CUDA_ERROR(hipFree(d_sv));
}