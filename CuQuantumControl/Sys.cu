#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "ApplyGates.hpp"
#include "QftRhoWise.hpp"
#include <cstring>
int runner()
{

    const int nIndexBits = 3;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;


    hipDoubleComplex h_sv[] = {{1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}, {8, 0}};
    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    // Initialize the values
    std::memcpy(d_sv, &h_sv, nSvSize * sizeof(hipDoubleComplex));
    
    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    ApplyQFTOnStateVector(d_sv, nIndexBits);

    for (int i = 0; i < nSvSize; i++)
    {
        std::cout << (d_sv[i].x) << "," << d_sv[i].y << " , " << static_cast<std::bitset<3>>(i) << std::endl;
    }
    std::cout << "\n";

    HANDLE_CUDA_ERROR(hipFree(d_sv));

    return hipSuccess;
}