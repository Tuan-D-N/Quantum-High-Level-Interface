#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <custatevec.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "../CudaControl/Helper.hpp"

#include "../functionality/randomArray.hpp"
#include "ApplySampler.hpp"

int sampleSV(custatevecHandle_t &handle,
             const int nIndexBits,
             const int bitOrdering[], // Qubits to measure
             const int bitStringLen,  // length of bitOrdering
             hipDoubleComplex d_sv[],
             custatevecIndex_t bitStrings_out[],
             int nShots,
             void *extraWorkspace,
             size_t &extraWorkspaceSizeInBytes,
             double randnums[])
{
    size_t extraWorkspaceSizeInBytes_CHECK = extraWorkspaceSizeInBytes;
    int nMaxShots = nShots;

    custatevecSamplerDescriptor_t sampler;
    // create sampler and check the size of external workspace
    CHECK_CUSTATEVECTOR(custatevecSamplerCreate(
        handle, d_sv, HIP_C_64F, nIndexBits, &sampler, nMaxShots,
        &extraWorkspaceSizeInBytes_CHECK));

    if (extraWorkspaceSizeInBytes_CHECK > extraWorkspaceSizeInBytes)
    {
        std::cout << "Extra space needed: " << extraWorkspaceSizeInBytes_CHECK - extraWorkspaceSizeInBytes << " Bytes";
        if (extraWorkspace != nullptr)
        {
            CHECK_CUDA(hipFree(extraWorkspace));
        }
        CHECK_CUDA(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes_CHECK));
    }

    // sample preprocess
    CHECK_CUSTATEVECTOR(custatevecSamplerPreprocess(
        handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes));

    // In real appliction, random numbers in range [0, 1) will be used.
    if (randnums == nullptr)
        randnums = new double[nShots];
        generateRandomArray(randnums, nShots);

    // sample bit strings
    CHECK_CUSTATEVECTOR(custatevecSamplerSample(
        handle, sampler, bitStrings_out, bitOrdering, bitStringLen, randnums, nShots,
        CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

    // destroy descriptor and handle
    CHECK_CUSTATEVECTOR(custatevecSamplerDestroy(sampler));
    extraWorkspaceSizeInBytes = extraWorkspaceSizeInBytes_CHECK;
    return EXIT_SUCCESS;
}