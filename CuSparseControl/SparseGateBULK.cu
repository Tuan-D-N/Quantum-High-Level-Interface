#include "hip/hip_runtime.h"
#include "SparseGateBULK.hpp"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>
#include <vector>
#include <algorithm>
#include "../CudaControl/Helper.hpp"
#include "SparseHelper.hpp"

// ---- helpers: bulk gather / scatter (column-major matrices: ld = d) ----
__global__ void bulk_gather_cols(
    hipDoubleComplex *__restrict__ M, int ld,      // M is d x B (column-major)
    const hipDoubleComplex *__restrict__ state_in, // full state vector
    const int *__restrict__ offsets,              // size d*B; column j then row r
    int d, int B)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = d * B;
    if (idx >= total)
        return;
    int col = idx / d;       // 0..B-1
    int row = idx - col * d; // 0..d-1
    int gidx = offsets[col * d + row];
    M[col * ld + row] = state_in[gidx];
}

__global__ void bulk_scatter_cols(
    hipDoubleComplex *__restrict__ state_out,       // full state vector
    const hipDoubleComplex *__restrict__ M, int ld, // Y matrix d x B (column-major)
    const int *__restrict__ offsets,               // size d*B
    int d, int B)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = d * B;
    if (idx >= total)
        return;
    int col = idx / d;
    int row = idx - col * d;
    int gidx = offsets[col * d + row];
    state_out[gidx] = M[col * ld + row];
}

// Apply sparse gate U to given qubits: optimized with bulk SpMM
int applySparseGateBulk(
    hipsparseHandle_t handle,
    int nQubits,
    const int *d_csrRowPtrU,
    const int *d_csrColIndU,
    const hipDoubleComplex *d_csrValU,
    const hipDoubleComplex *d_state_in,
    hipDoubleComplex *d_state_out,
    const std::vector<int> &targetQubits,
    const std::vector<int> &controlQubits,
    int nnzU)
{
    const int k = static_cast<int>(targetQubits.size());
    const int d = 1 << k;
    const int dim = 1 << nQubits;

    // (Optional but robust) pass-through initialization for blocks that won't be touched.
    // If you want exact parity with your scalar path, do the same there or remove this.
    if (d_state_out != d_state_in)
    {
        CHECK_CUDA(hipMemcpy((void *)d_state_out, (const void *)d_state_in,
                              sizeof(hipDoubleComplex) * dim, hipMemcpyDeviceToDevice));
    }

    // --- Sparse descriptor for U (d x d) ---
    hipsparseSpMatDescr_t matU;
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matU, d, d, nnzU,
        (void *)d_csrRowPtrU, (void *)d_csrColIndU, (void *)d_csrValU,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F));

    const hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    const hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);

    // --- Build non-target list & control mask ---
    std::vector<int> nonTargetQubits;
    nonTargetQubits.reserve(nQubits - k);
    for (int q = 0; q < nQubits; ++q)
        if (std::find(targetQubits.begin(), targetQubits.end(), q) == targetQubits.end())
            nonTargetQubits.push_back(q);

    int controlMask = 0;
    for (int cq : controlQubits)
        controlMask |= (1 << cq);

    // --- Enumerate eligible blocks (controls satisfied) ---
    const int nBlocks = 1 << (nQubits - k);
    std::vector<int> workBlocks;
    workBlocks.reserve(nBlocks);
    for (int blk = 0; blk < nBlocks; ++blk)
    {
        int nonTargetMask = 0;
        for (size_t i = 0; i < nonTargetQubits.size(); ++i)
            if ((blk >> i) & 1)
                nonTargetMask |= (1 << nonTargetQubits[i]);
        if ((nonTargetMask & controlMask) == controlMask)
            workBlocks.push_back(blk);
    }
    if (workBlocks.empty())
    {
        hipsparseDestroySpMat(matU);
        return 0;
    }

    // --- Buffers / descriptors reused across batches ---
    hipDoubleComplex *d_X = nullptr, *d_Y = nullptr;
    int *d_offsets = nullptr;
    void *dBuffer = nullptr;
    size_t dBufferSize = 0;

    // Dense descriptors (will be recreated if B changes)
    hipsparseDnMatDescr_t matX = nullptr, matY = nullptr;
    int currentB = 0; // track current descriptor column count

    auto destroyDnMats = [&]()
    {
        if (matX)
        {
            hipsparseDestroyDnMat(matX);
            matX = nullptr;
        }
        if (matY)
        {
            hipsparseDestroyDnMat(matY);
            matY = nullptr;
        }
    };

    // Simple soft cap heuristic
    const size_t capBytes = size_t(64) << 20; // ~64 MiB
    auto maxBByMem = [&](size_t wsBytes)
    {
        // per column B: X(d) + Y(d) complexes + offsets(d) ints
        // 16 bytes per hipDoubleComplex, 4 bytes per int
        double perB = double(d) * 16.0 + double(d) * 16.0 + double(d) * 4.0;
        double avail = double(capBytes > wsBytes ? (capBytes - wsBytes) : 0);
        int B = (perB > 0.0) ? int(avail / perB) : 1;
        return std::max(1, B);
    };

    const int totalW = static_cast<int>(workBlocks.size());
    int processed = 0;

    while (processed < totalW)
    {
        // ---- 1) Start with a guess for B ----
        int remaining = totalW - processed;
        int Bguess = std::min(remaining, std::max(1, 64 / std::max(1, d / 64)));

        // Create temp descriptors for query at Bguess (if needed)
        if (!matX || currentB != Bguess)
        {
            destroyDnMats();
            CHECK_CUSPARSE(hipsparseCreateDnMat(&matX, d, Bguess, d, nullptr, HIP_C_64F, HIPSPARSE_ORDER_COL));
            CHECK_CUSPARSE(hipsparseCreateDnMat(&matY, d, Bguess, d, nullptr, HIP_C_64F, HIPSPARSE_ORDER_COL));
            currentB = Bguess;
        }

        // Query workspace for this Bguess
        size_t needed_guess = 0;
        CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matU, matX, &beta, matY, HIP_C_64F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &needed_guess));

        // Cap B by memory
        int B = std::min(remaining, std::min(maxBByMem(needed_guess), Bguess));
        if (B < 1)
            B = 1;

        // If B changed from the guess, recreate descriptors and re-query workspace for the FINAL B
        if (B != currentB)
        {
            destroyDnMats();
            CHECK_CUSPARSE(hipsparseCreateDnMat(&matX, d, B, d, nullptr, HIP_C_64F, HIPSPARSE_ORDER_COL));
            CHECK_CUSPARSE(hipsparseCreateDnMat(&matY, d, B, d, nullptr, HIP_C_64F, HIPSPARSE_ORDER_COL));
            currentB = B;
        }
        size_t needed = 0;
        CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matU, matX, &beta, matY, HIP_C_64F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &needed));

        // Allocate/reallocate for final B
        const size_t bytesX = size_t(d) * B * sizeof(hipDoubleComplex);
        const size_t bytesY = size_t(d) * B * sizeof(hipDoubleComplex);
        const size_t bytesO = size_t(d) * B * sizeof(int);

        auto reallocIfNeeded = [](void **ptr, size_t bytes)
        {
            if (*ptr)
                hipFree(*ptr);
            CHECK_CUDA(hipMalloc(ptr, bytes));
            return static_cast<int>(hipSuccess);
        };
        if (!d_X)
            reallocIfNeeded((void **)&d_X, bytesX);
        else
        {
            hipFree(d_X);
            CHECK_CUDA(hipMalloc((void **)&d_X, bytesX));
        }
        if (!d_Y)
            reallocIfNeeded((void **)&d_Y, bytesY);
        else
        {
            hipFree(d_Y);
            CHECK_CUDA(hipMalloc((void **)&d_Y, bytesY));
        }
        if (!d_offsets)
            reallocIfNeeded((void **)&d_offsets, bytesO);
        else
        {
            hipFree(d_offsets);
            CHECK_CUDA(hipMalloc((void **)&d_offsets, bytesO));
        }

        if (!dBuffer || dBufferSize < needed)
        {
            if (dBuffer)
                hipFree(dBuffer);
            CHECK_CUDA(hipMalloc(&dBuffer, needed));
            dBufferSize = needed;
        }

        // Bind values to DnMats
        CHECK_CUSPARSE(hipsparseDnMatSetValues(matX, d_X));
        CHECK_CUSPARSE(hipsparseDnMatSetValues(matY, d_Y));

        // ---- 2) Build offsets for this batch ----
        std::vector<int> h_offsets(size_t(d) * B);
        for (int j = 0; j < B; ++j)
        {
            int blk = workBlocks[processed + j];
            int nonTargetMask = 0;
            for (size_t i = 0; i < nonTargetQubits.size(); ++i)
                if ((blk >> i) & 1)
                    nonTargetMask |= (1 << nonTargetQubits[i]);
            for (int b = 0; b < d; ++b)
            {
                int targetMask = 0;
                for (int q = 0; q < k; ++q)
                    if ((b >> q) & 1)
                        targetMask |= (1 << targetQubits[q]);
                h_offsets[j * d + b] = nonTargetMask | targetMask; // column-major fill
            }
        }
        CHECK_CUDA(hipMemcpy(d_offsets, h_offsets.data(), bytesO, hipMemcpyHostToDevice));

        // ---- 3) Bulk gather X (d x B, column-major) ----
        {
            int threads = 256;
            int blocks = int((size_t(d) * B + threads - 1) / threads);
            bulk_gather_cols CUDA_KERNEL(blocks, threads)(d_X, d, d_state_in, d_offsets, d, B);
            CHECK_CUDA(hipGetLastError());
        }

        // ---- 4) SpMM: Y = U * X ----
        CHECK_CUSPARSE(hipsparseSpMM(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matU, matX, &beta, matY, HIP_C_64F,
            HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));

        // ---- 5) Bulk scatter Y back ----
        {
            int threads = 256;
            int blocks = int((size_t(d) * B + threads - 1) / threads);
            bulk_scatter_cols CUDA_KERNEL(blocks, threads)(d_state_out, d_Y, d, d_offsets, d, B);
            CHECK_CUDA(hipGetLastError());
        }

        processed += B;
    }

    // Cleanup
    if (d_X)
        hipFree(d_X);
    if (d_Y)
        hipFree(d_Y);
    if (d_offsets)
        hipFree(d_offsets);
    if (dBuffer)
        hipFree(dBuffer);
    if (matX)
        hipsparseDestroyDnMat(matX);
    if (matY)
        hipsparseDestroyDnMat(matY);
    hipsparseDestroySpMat(matU);
    return 0;
}
