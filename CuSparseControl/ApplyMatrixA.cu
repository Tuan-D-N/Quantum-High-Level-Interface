#include "ApplyMatrixA.hpp"
#include <stdlib.h>           // EXIT_FAILURE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>
#include <hipsparse.h>
#include "../CudaControl/Helper.hpp"
#include "../functionality/WriteAdjMat.hpp"

// theta slow, r fast
int applyInterpolationMatrix(int evenqubits, hipDoubleComplex *rThetaVector, hipDoubleComplex *&xyVector)
{
    // Host problem definition
    int A_num_rows = 1 << evenqubits;
    int A_num_cols = 1 << evenqubits;
    int A_max_nnz = 4 * A_num_rows;

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    //--------------------------------------------------------------------------
    // Device memory management
    int *dA_csrOffsets, *dA_columns;
    hipDoubleComplex *dA_values;
    CHECK_CUDA(hipMallocManaged((void **)&dA_csrOffsets,
                                 (A_num_rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_columns, A_max_nnz * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_values, A_max_nnz * sizeof(hipDoubleComplex)))

    int postIndexSize, postOffsetSize, postValueSize;

    // Unified Memory Cuda Write
    writeMatAMiniCSC(dA_csrOffsets, dA_columns, dA_values, evenqubits, postOffsetSize, postIndexSize, postValueSize, false);

    // Vector
    if (xyVector == nullptr)
    {
        CHECK_CUDA(hipMallocManaged((void **)&xyVector, A_num_cols * sizeof(hipDoubleComplex)));
        for (int i = 0; i < A_num_cols; ++i)
        {
            xyVector[i] = {0, 0};
        }
    }

    //--------------------------------------------------------------------------
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, postValueSize,
                                     dA_csrOffsets, dA_columns, dA_values,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F))

    hipsparseDnVecDescr_t vectorIn;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorIn, A_num_rows, rThetaVector, HIP_C_64F));

    hipsparseDnVecDescr_t vectorOut;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorOut, A_num_rows, xyVector, HIP_C_64F));

    //---------------------------------------------------------------------------

    // Workspace buffer
    void *dBuffer = nullptr;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                                           &alpha, matA, vectorIn, &beta, vectorOut,
                                           HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // Perform the SpMV operation
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                                &alpha, matA, vectorIn, &beta, vectorOut,
                                HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vectorIn);
    hipsparseDestroyDnVec(vectorOut);

    CHECK_CUDA(hipFree(dA_csrOffsets))
    CHECK_CUDA(hipFree(dA_columns))
    CHECK_CUDA(hipFree(dA_values))

    return hipSuccess;
}