#include "ApplySparseCSRMat.hpp"

int applySparseCSRMat(hipsparseHandle_t handle,
                      std::span<int> csrOffsets,
                      std::span<int> csrRows,
                      std::span<hipDoubleComplex> values,
                      std::span<hipDoubleComplex> svInput,
                      std::span<hipDoubleComplex> svOutput)
{
    hipsparseSpMatDescr_t matrixOBJ;
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    int matrix_num_rows = svInput.size();
    int matrix_num_cols = svInput.size();

    CHECK_CUSPARSE(hipsparseCreateCsr(&matrixOBJ, matrix_num_rows, matrix_num_cols, values.size(),
                                     csrOffsets.data(), csrRows.data(), values.data(),
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F))

    hipsparseDnVecDescr_t vectorIn;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorIn, svInput.size(), svInput.data(), HIP_C_64F));

    hipsparseDnVecDescr_t vectorOut;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorOut, svOutput.size(), svOutput.data(), HIP_C_64F));

    //---------------------------------------------------------------------------

    // Workspace buffer
    void *dBuffer = nullptr;

    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                                           &alpha, matrixOBJ, vectorIn, &beta, vectorOut,
                                           HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // Perform the SpMV operation
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                                &alpha, matrixOBJ, vectorIn, &beta, vectorOut,
                                HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

    hipsparseDestroySpMat(matrixOBJ);
    hipsparseDestroyDnVec(vectorIn);
    hipsparseDestroyDnVec(vectorOut);
    
    return hipSuccess;
}