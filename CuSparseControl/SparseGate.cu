#include "hip/hip_runtime.h"
#include "SparseGate.hpp"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>
#include <vector>
#include <algorithm>
#include "../CudaControl/Helper.hpp"

#ifdef __HIPCC__
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#else
#define CUDA_KERNEL(...) 
#endif

// The gather and scatter kernels are still necessary for correctness and performance.
__global__ void gather_kernel(hipDoubleComplex* d_out, const hipDoubleComplex* d_in, const int* d_offsets, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        d_out[i] = d_in[d_offsets[i]];
    }
}

__global__ void scatter_kernel(hipDoubleComplex* d_out, const hipDoubleComplex* d_in, const int* d_offsets, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        d_out[d_offsets[i]] = d_in[i];
    }
}

// Apply sparse gate U to given qubits (q0, q1, ..., qk).
// - nQubits: total system qubits
// - d_csrRowPtrU, d_csrColIndU, d_csrValU: CSR of U (size d x d, where d=2^k)
// - d_state_in/out: device statevectors, length 2^n
// - targetQubits: list of k target qubits (ascending order)
// Apply sparse gate U to given qubits (q0, q1, ..., qk)
int applySparseGate(
    hipsparseHandle_t handle,
    int nQubits,
    const int* d_csrRowPtrU,
    const int* d_csrColIndU,
    const hipDoubleComplex* d_csrValU,
    const hipDoubleComplex* d_state_in,
    hipDoubleComplex* d_state_out,
    const std::vector<int>& targetQubits,
    int nnzU)
{
    int k = targetQubits.size();
    int d = 1 << k;
    int dim = 1 << nQubits;

    hipsparseSpMatDescr_t matU;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matU, d, d, nnzU, (void*)d_csrRowPtrU, (void*)d_csrColIndU, (void*)d_csrValU,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F));

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    
    hipDoubleComplex* d_block_in;
    hipDoubleComplex* d_block_out;
    CHECK_CUDA(hipMalloc(&d_block_in, d * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_block_out, d * sizeof(hipDoubleComplex)));

    hipsparseDnVecDescr_t vecX, vecY;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, d, d_block_in, HIP_C_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, d, d_block_out, HIP_C_64F));

    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matU, vecX, &beta, vecY,
                                           HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    void* dBuffer = nullptr;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    int nBlocks = 1 << (nQubits - k);
    std::vector<int> nonTargetQubits;
    for (int q = 0; q < nQubits; ++q) {
        if (std::find(targetQubits.begin(), targetQubits.end(), q) == targetQubits.end()) {
            nonTargetQubits.push_back(q);
        }
    }

    std::vector<int> h_offsets(d);
    int* d_offsets;
    CHECK_CUDA(hipMalloc(&d_offsets, d * sizeof(int)));

    for (int blk = 0; blk < nBlocks; ++blk) {
        int nonTargetMask = 0;
        for (size_t i = 0; i < nonTargetQubits.size(); ++i) {
            if ((blk >> i) & 1) {
                nonTargetMask |= (1 << nonTargetQubits[i]);
            }
        }

        for (int b = 0; b < d; ++b) {
            int targetMask = 0;
            for (int q = 0; q < k; ++q) {
                if ((b >> q) & 1) {
                    targetMask |= (1 << targetQubits[q]);
                }
            }
            h_offsets[b] = nonTargetMask | targetMask;
        }

        CHECK_CUDA(hipMemcpy(d_offsets, h_offsets.data(), d * sizeof(int), hipMemcpyHostToDevice));

        int threads_per_block = 256;
        int blocks = (d + threads_per_block - 1) / threads_per_block;
        gather_kernel CUDA_KERNEL(blocks, threads_per_block) (d_block_in, d_state_in, d_offsets, d);
        CHECK_CUDA(hipPeekAtLastError());
        hipDeviceSynchronize();

        CHECK_CUDA(hipMemset(d_block_out, 0, d * sizeof(hipDoubleComplex)));

        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matU, vecX, &beta, vecY,
                                    HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
        CHECK_CUDA(hipDeviceSynchronize());

        scatter_kernel CUDA_KERNEL(blocks, threads_per_block) (d_state_out, d_block_out, d_offsets, d);
        CHECK_CUDA(hipPeekAtLastError());
        hipDeviceSynchronize();
    }

    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_block_in));
    CHECK_CUDA(hipFree(d_block_out));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matU));

    return 0;
}