#include "SparseGate.hpp"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>
#include <vector>
#include <algorithm>
#include "../CudaControl/Helper.hpp"

// Apply sparse gate U to given qubits (q0, q1, ..., qk).
// - nQubits: total system qubits
// - d_csrRowPtrU, d_csrColIndU, d_csrValU: CSR of U (size d x d, where d=2^k)
// - d_state_in/out: device statevectors, length 2^n
// - targetQubits: list of k target qubits (ascending order)
// Apply sparse gate U to given qubits (q0, q1, ..., qk)
int applySparseGate(
    hipsparseHandle_t handle,
    int nQubits,
    const int* d_csrRowPtrU,
    const int* d_csrColIndU,
    const hipDoubleComplex* d_csrValU,
    const hipDoubleComplex* d_state_in,
    hipDoubleComplex* d_state_out,
    const std::vector<int>& targetQubits,
    int nnzU)
{
    int k = targetQubits.size();       // number of qubits this gate acts on
    int d = 1 << k;                    // dimension of U
    int dim = 1 << nQubits;            // full state dimension

    // --- Build sparse matrix descriptor
    hipsparseSpMatDescr_t matU;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matU,
        d, d, nnzU,
        (void*)d_csrRowPtrU,
        (void*)d_csrColIndU,
        (void*)d_csrValU,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_C_64F));

    // Work buffer (reused across blocks)
    size_t bufferSize = 0;
    void* dBuffer = nullptr;
    double alpha = 1.0, beta = 0.0;

    // Temporary dense descriptors
    hipsparseDnVecDescr_t vecX, vecY;

    // Number of blocks of length d
    int nBlocks = 1 << (nQubits - k);

    // Loop over blocks
    for (int blk = 0; blk < nBlocks; blk++) {
        // --- Compute base index for this block
        int base = 0;
        int tmp = blk;
        for (int q = 0; q < nQubits; q++) {
            if (std::find(targetQubits.begin(), targetQubits.end(), q) != targetQubits.end())
                continue; // skip target qubits
            int bit = tmp & 1;
            tmp >>= 1;
            base |= (bit << q);
        }

        // --- Collect d amplitudes for all combinations of target qubits
        std::vector<int> offsets(d, 0);
        for (int b = 0; b < d; b++) {
            int idx = base;
            for (int q = 0; q < k; q++) {
                if (b & (1 << q)) idx |= (1 << targetQubits[q]);
            }
            offsets[b] = idx;
        }

        // --- Device dense views
        hipDoubleComplex* d_block_in  = (hipDoubleComplex*)(d_state_in  + offsets[0]);
        hipDoubleComplex* d_block_out = (hipDoubleComplex*)(d_state_out + offsets[0]);

        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, d, d_block_in, HIP_C_64F));
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, d, d_block_out, HIP_C_64F));

        // Query buffer size (once is enough, but we keep it simple here)
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matU, vecX, &beta, vecY,
            HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));

        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

        // SpMV: y = U * x
        CHECK_CUSPARSE(hipsparseSpMV(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matU, vecX, &beta, vecY,
            HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

        CHECK_CUDA(hipFree(dBuffer));
        CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
        CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    }

    CHECK_CUSPARSE(hipsparseDestroySpMat(matU));

    return 0;
}
