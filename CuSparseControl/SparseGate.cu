#include "SparseGate.hpp"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>
#include <vector>
#include <algorithm>
#include "../CudaControl/Helper.hpp"
#include "SparseHelper.hpp"

// Apply sparse gate U to given qubits (q0, q1, ..., qk).
// - nQubits: total system qubits
// - d_csrRowPtrU, d_csrColIndU, d_csrValU: CSR of U (size d x d, where d=2^k)
// - d_state_in/out: device statevectors, length 2^n
// - targetQubits: list of k target qubits (ascending order)
// Apply sparse gate U to given qubits (q0, q1, ..., qk)
int applySparseGate(
    hipsparseHandle_t handle,
    int nQubits,
    const int *d_csrRowPtrU,
    const int *d_csrColIndU,
    const hipDoubleComplex *d_csrValU,
    const hipDoubleComplex *d_state_in,
    hipDoubleComplex *d_state_out,
    const std::vector<int> &targetQubits,
    int nnzU)
{
    // k is the number of target qubits (dimension of the gate U)
    int k = targetQubits.size();
    // d is the dimension of the sub-block/gate U (d = 2^k)
    int d = 1 << k;
    // dim is the total size of the state vector (dim = 2^nQubits)
    int dim = 1 << nQubits;

    // --- 1. Setup cuSPARSE Matrix Descriptor for the Gate U ---
    hipsparseSpMatDescr_t matU;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matU, d, d, nnzU, (void *)d_csrRowPtrU, (void *)d_csrColIndU, (void *)d_csrValU,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F));

    // Define SpMV scalars: alpha=1.0, beta=0.0 (Operation: Y = 1.0 * U * X + 0.0 * Y)
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);

    // --- 2. Allocate Scratchpad Memory for Sub-Blocks ---
    hipDoubleComplex *d_block_in;
    hipDoubleComplex *d_block_out;
    // Allocate device memory for the input and output sub-vectors (size d)
    CHECK_CUDA(hipMalloc(&d_block_in, d * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_block_out, d * sizeof(hipDoubleComplex)));

    // --- 3. Setup cuSPARSE Dense Vector Descriptors ---
    hipsparseDnVecDescr_t vecX, vecY;
    // Descriptor for input sub-vector X
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, d, d_block_in, HIP_C_64F));
    // Descriptor for output sub-vector Y
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, d, d_block_out, HIP_C_64F));

    // --- 4. Query and Allocate cuSPARSE Workspace Buffer ---
    size_t bufferSize = 0;
    // Determine the required buffer size for the SpMV operation
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matU, vecX, &beta, vecY,
                                           HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    void *dBuffer = nullptr;
    // Allocate the buffer
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // --- 5. Prepare for Block Iteration ---
    // nBlocks is the number of sub-blocks (size d) the state vector is partitioned into (2^(n-k))
    int nBlocks = 1 << (nQubits - k);

    // Identify qubits NOT in the target list (these define the block index 'blk')
    std::vector<int> nonTargetQubits;
    for (int q = 0; q < nQubits; ++q)
    {
        if (std::find(targetQubits.begin(), targetQubits.end(), q) == targetQubits.end())
        {
            nonTargetQubits.push_back(q);
        }
    }

    // Allocate host and device memory for the state vector offsets
    std::vector<int> h_offsets(d);
    int *d_offsets;
    CHECK_CUDA(hipMalloc(&d_offsets, d * sizeof(int)));

    // --- 6. Iterate through all computational blocks (Non-Target Qubit States) ---
    for (int blk = 0; blk < nBlocks; ++blk)
    {
        // --- A. Calculate the base offset/mask for the current block ---
        int nonTargetMask = 0;
        // The 'blk' loop index represents the state of the non-target qubits
        for (size_t i = 0; i < nonTargetQubits.size(); ++i)
        {
            if ((blk >> i) & 1)
            {
                nonTargetMask |= (1 << nonTargetQubits[i]);
            }
        }

        // --- B. Calculate full state vector indices for the current block ---
        // Iterate through all 'd' indices within the current block
        for (int b = 0; b < d; ++b)
        {
            int targetMask = 0;
            // The 'b' index represents the state of the target qubits
            for (int q = 0; q < k; ++q)
            {
                if ((b >> q) & 1)
                {
                    targetMask |= (1 << targetQubits[q]);
                }
            }
            // The full index is the sum of the non-target base mask and the target mask
            h_offsets[b] = nonTargetMask | targetMask;
        }

        // 1. Transfer offsets to device
        CHECK_CUDA(hipMemcpy(d_offsets, h_offsets.data(), d * sizeof(int), hipMemcpyHostToDevice));

        // 2. Gather: Read input sub-vector from global state into scratchpad
        int threads_per_block = 256;
        int blocks = (d + threads_per_block - 1) / threads_per_block;
        gather_kernel CUDA_KERNEL(blocks, threads_per_block)(d_block_in, d_state_in, d_offsets, d);
        CHECK_CUDA(hipPeekAtLastError());
        hipDeviceSynchronize();

        // 3. Apply: Perform the Sparse Matrix-Vector Multiplication (SpMV)
        CHECK_CUDA(hipMemset(d_block_out, 0, d * sizeof(hipDoubleComplex))); // Clear output block
        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matU, vecX, &beta, vecY,
                                    HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
        CHECK_CUDA(hipDeviceSynchronize());

        // 4. Scatter: Write output sub-vector from scratchpad back to global state
        scatter_kernel CUDA_KERNEL(blocks, threads_per_block)(d_state_out, d_block_out, d_offsets, d);
        CHECK_CUDA(hipPeekAtLastError());
        hipDeviceSynchronize();
    }

    // --- 7. Cleanup Resources ---
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_block_in));
    CHECK_CUDA(hipFree(d_block_out));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matU));

    return 0;
}

// Apply sparse gate U to given qubits (q0, q1, ..., qk).
// - nQubits: total system qubits
// - d_csrRowPtrU, d_csrColIndU, d_csrValU: CSR of U (size d x d, where d=2^k)
// - d_state_in/out: device statevectors, length 2^n
// - targetQubits: list of k target qubits (ascending order)
// - controlQubits: list of k control qubits (ascending order)
// Apply sparse gate U to given qubits (q0, q1, ..., qk)
int applySparseGate(
    hipsparseHandle_t handle,
    int nQubits,
    const int *d_csrRowPtrU,
    const int *d_csrColIndU,
    const hipDoubleComplex *d_csrValU,
    const hipDoubleComplex *d_state_in,
    hipDoubleComplex *d_state_out,
    const std::vector<int> &targetQubits,
    const std::vector<int> &controlQubits,
    int nnzU)
{
    // k is the number of target qubits (dimension of the gate U)
    int k = targetQubits.size();
    // d is the dimension of the sub-block/gate U (d = 2^k)
    int d = 1 << k;
    // dim is the total size of the state vector (dim = 2^nQubits)
    int dim = 1 << nQubits;

    // --- 1. Setup cuSPARSE Matrix Descriptor for the Gate U ---
    hipsparseSpMatDescr_t matU;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matU, d, d, nnzU, (void *)d_csrRowPtrU, (void *)d_csrColIndU, (void *)d_csrValU,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F));

    // Define SpMV scalars: alpha=1.0, beta=0.0 (Operation: Y = 1.0 * U * X + 0.0 * Y)
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);

    // --- 2. Allocate Scratchpad Memory for Sub-Blocks ---
    hipDoubleComplex *d_block_in;
    hipDoubleComplex *d_block_out;
    // Allocate device memory for the input and output sub-vectors (size d)
    CHECK_CUDA(hipMalloc(&d_block_in, d * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_block_out, d * sizeof(hipDoubleComplex)));

    // --- 3. Setup cuSPARSE Dense Vector Descriptors ---
    hipsparseDnVecDescr_t vecX, vecY;
    // Descriptor for input sub-vector X
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, d, d_block_in, HIP_C_64F));
    // Descriptor for output sub-vector Y
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, d, d_block_out, HIP_C_64F));

    // --- 4. Query and Allocate cuSPARSE Workspace Buffer ---
    size_t bufferSize = 0;
    // Determine the required buffer size for the SpMV operation
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matU, vecX, &beta, vecY,
                                           HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    void *dBuffer = nullptr;
    // Allocate the buffer
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // --- 5. Prepare for Block Iteration ---
    // nBlocks is the number of sub-blocks (size d) the state vector is partitioned into (2^(n-k))
    int nBlocks = 1 << (nQubits - k);

    // Identify qubits NOT in the target list (these define the block index 'blk')
    std::vector<int> nonTargetQubits;
    for (int q = 0; q < nQubits; ++q)
    {
        if (std::find(targetQubits.begin(), targetQubits.end(), q) == targetQubits.end())
        {
            nonTargetQubits.push_back(q);
        }
    }

    // Compute control mask for the qubits that must be '1'
    int controlMask = 0;
    for (int cq : controlQubits)
        controlMask |= (1 << cq);

    // Allocate host and device memory for the state vector offsets
    std::vector<int> h_offsets(d);
    int *d_offsets;
    CHECK_CUDA(hipMalloc(&d_offsets, d * sizeof(int)));

    // --- 6. Iterate through all computational blocks (Non-Target Qubit States) ---
    for (int blk = 0; blk < nBlocks; ++blk)
    {
        // --- A. Calculate the base offset/mask for the current block ---
        int nonTargetMask = 0;
        // The 'blk' loop index represents the state of the non-target qubits
        for (size_t i = 0; i < nonTargetQubits.size(); ++i)
        {
            if ((blk >> i) & 1)
            {
                nonTargetMask |= (1 << nonTargetQubits[i]);
            }
        }

        // ✅ Skip this block if control qubits not all active
        if ((nonTargetMask & controlMask) != controlMask)
            continue;

        // --- B. Calculate full state vector indices for the current block ---
        // Iterate through all 'd' indices within the current block
        for (int b = 0; b < d; ++b)
        {
            int targetMask = 0;
            // The 'b' index represents the state of the target qubits
            for (int q = 0; q < k; ++q)
            {
                if ((b >> q) & 1)
                {
                    targetMask |= (1 << targetQubits[q]);
                }
            }
            // The full index is the sum of the non-target base mask and the target mask
            h_offsets[b] = nonTargetMask | targetMask;
        }

        // 1. Transfer offsets to device
        CHECK_CUDA(hipMemcpy(d_offsets, h_offsets.data(), d * sizeof(int), hipMemcpyHostToDevice));

        // 2. Gather: Read input sub-vector from global state into scratchpad
        int threads_per_block = 256;
        int blocks = (d + threads_per_block - 1) / threads_per_block;
        gather_kernel CUDA_KERNEL(blocks, threads_per_block)(d_block_in, d_state_in, d_offsets, d);
        CHECK_CUDA(hipPeekAtLastError());
        hipDeviceSynchronize();

        // 3. Apply: Perform the Sparse Matrix-Vector Multiplication (SpMV)
        CHECK_CUDA(hipMemset(d_block_out, 0, d * sizeof(hipDoubleComplex))); // Clear output block
        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matU, vecX, &beta, vecY,
                                    HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
        CHECK_CUDA(hipDeviceSynchronize());

        // 4. Scatter: Write output sub-vector from scratchpad back to global state
        scatter_kernel CUDA_KERNEL(blocks, threads_per_block)(d_state_out, d_block_out, d_offsets, d);
        CHECK_CUDA(hipPeekAtLastError());
        hipDeviceSynchronize();
    }

    // --- 7. Cleanup Resources ---
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_block_in));
    CHECK_CUDA(hipFree(d_block_out));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matU));

    return 0;
}