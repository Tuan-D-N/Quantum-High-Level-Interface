#include "hip/hip_runtime.h"
#include "ChebyshevMatrixExponentialCoeff.hpp"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <cmath>
#include <span>
#include <optional>
#include <>
#include "../CudaControl/Helper.hpp"   // CHECK_CUDA / CHECK_CUSPARSE
#include "SparseHelper.hpp"            // gather_kernel / scatter_kernel

// ==========================================================
// Device kernels
// ==========================================================

// out[i] += coeff * tmp[i]
__global__ void axpy_scale_kernel_complex(hipDoubleComplex* out,
                                          const hipDoubleComplex* tmp,
                                          hipDoubleComplex coeff,
                                          int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    hipDoubleComplex term = hipCmul(coeff, tmp[i]);
    out[i] = hipCadd(out[i], term);
}

// out[i] = coeff * in[i]
__global__ void scale_kernel_complex(hipDoubleComplex* out,
                                     const hipDoubleComplex* in,
                                     hipDoubleComplex coeff,
                                     int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    out[i] = hipCmul(coeff, in[i]);
}


// ==========================================================
// exp(iA) v ≈ Σ_{k=0}^order gamma[k] A^k v     (DEVICE CSR, raw ptrs)
// ==========================================================
int expiAv_chebyshev_gamma_cusparse_device(
    hipsparseHandle_t handle,
    int n, int nnz, int order,
    const int* d_csrRowPtr,                  // DEVICE pointers (no span)
    const int* d_csrColInd,                  // DEVICE pointers (no span)
    const hipDoubleComplex* d_csrVal,         // DEVICE pointers (no span)
    const std::vector<hipDoubleComplex> gamma,
    hipDoubleComplex* d_v_in_out)
{
    if (order < 0) return 0;
    if (!d_csrRowPtr || !d_csrColInd || !d_csrVal || !d_v_in_out) return -1;
    if ((int)gamma.size() < order + 1) return -2;

    // cuSPARSE descriptors
    const hipDoubleComplex one  = make_hipDoubleComplex(1.0, 0.0);
    const hipDoubleComplex zero = make_hipDoubleComplex(0.0, 0.0);

    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, n, n, nnz,
        (void*)d_csrRowPtr, (void*)d_csrColInd, (void*)d_csrVal,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F));

    hipDoubleComplex *d_tmp_in = nullptr, *d_tmp_out = nullptr;
    CHECK_CUDA(hipMalloc(&d_tmp_in,  n * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_tmp_out, n * sizeof(hipDoubleComplex)));
    // tmp_in <- v (we keep d_v_in_out as the accumulator/output)
    CHECK_CUDA(hipMemcpy(d_tmp_in, d_v_in_out, n*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice));

    hipsparseDnVecDescr_t vecX, vecY;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, d_tmp_in,  HIP_C_64F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, n, d_tmp_out, HIP_C_64F));

    size_t bufferSize = 0;
    void*  dBuffer = nullptr;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &one, matA, vecX, &zero, vecY,
        HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    const int threads = 256;
    const int blocks  = (n + threads - 1) / threads;

    // out = gamma[0] * v
    scale_kernel_complex CUDA_KERNEL(blocks, threads) (d_v_in_out, d_tmp_in, gamma[0], n);
    CHECK_CUDA(hipGetLastError());

    // Power-series evaluation:
    // tmp_out = A * tmp_in; out += gamma[i] * tmp_out; swap(tmp_in,tmp_out)
    for (int i = 1; i <= order; ++i) {
        CHECK_CUSPARSE(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &one, matA, vecX, &zero, vecY,
            HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

        axpy_scale_kernel_complex CUDA_KERNEL(blocks, threads) (d_v_in_out, d_tmp_out, gamma[i], n);
        CHECK_CUDA(hipGetLastError());

        std::swap(d_tmp_in, d_tmp_out);
        CHECK_CUSPARSE(hipsparseDnVecSetValues(vecX, d_tmp_in));
        CHECK_CUSPARSE(hipsparseDnVecSetValues(vecY, d_tmp_out));
    }

    // Cleanup
    hipFree(dBuffer);
    hipFree(d_tmp_in);
    hipFree(d_tmp_out);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroySpMat(matA);
    return 0;
}

// ==========================================================
// exp(iA) v — HOST CSR spans -> (copy CSR to device) -> device eval
// ==========================================================
int expiAv_chebyshev_gamma_cusparse_host(
    hipsparseHandle_t handle,
    int n, int nnz, int order,
    std::span<const int> h_csrRowPtr,
    std::span<const int> h_csrColInd,
    std::span<const hipDoubleComplex> h_csrVal,
    hipDoubleComplex* d_v_in_out,
    const double t /* = +1 for exp(-iA), -1 for exp(+iA) */)
{
    if (order < 0) return 0;
    if ((int)h_csrRowPtr.size() != n + 1 || (int)h_csrColInd.size() != nnz || (int)h_csrVal.size() != nnz)
        return -1;

    // gamma from host CSR (spectral-only scaling; auto β = ||A||_1 if not provided)
    std::vector<hipDoubleComplex> gamma =
        chebyshev_exp_gamma_spectral_csr(n, h_csrRowPtr, h_csrColInd, h_csrVal, t, order, std::nullopt);

    // Copy CSR to device
    int *d_r = nullptr, *d_c = nullptr;
    hipDoubleComplex* d_v = nullptr;
    CHECK_CUDA(hipMalloc(&d_r, sizeof(int)*(n+1)));
    CHECK_CUDA(hipMalloc(&d_c, sizeof(int)*nnz));
    CHECK_CUDA(hipMalloc(&d_v, sizeof(hipDoubleComplex)*nnz));
    CHECK_CUDA(hipMemcpy(d_r, h_csrRowPtr.data(), sizeof(int)*(n+1), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_c, h_csrColInd.data(), sizeof(int)*nnz,  hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_v, h_csrVal.data(),    sizeof(hipDoubleComplex)*nnz, hipMemcpyHostToDevice));

    // Apply on device
    int rc = expiAv_chebyshev_gamma_cusparse_device(
        handle, n, nnz, order,
        d_r, d_c, d_v,
        gamma,
        d_v_in_out);

    // Free temps
    hipFree(d_r);
    hipFree(d_c);
    hipFree(d_v);
    return rc;
}

// ==========================================================
// Controlled exp(iA) on a target subspace — DEVICE CSR + gamma
// ==========================================================
int applyControlledExpChebyshev_cusparse_device(
    hipsparseHandle_t handle,
    int nQubits,
    const int* d_csrRowPtr,                  // DEVICE CSR of the target operator (dim d)
    const int* d_csrColInd,
    const hipDoubleComplex* d_csrVal,
    const std::vector<hipDoubleComplex> gamma,
    hipDoubleComplex* d_state,
    const std::vector<int>& targetQubits,
    const std::vector<int>& controlQubits,
    int nnz,
    int order)
{
    const int k   = static_cast<int>(targetQubits.size());
    const int d   = 1 << k;
    const int nBlocks = 1 << (nQubits - k);

    // Non-target set
    std::vector<int> nonTarget;
    nonTarget.reserve(nQubits - k);
    for (int q = 0; q < nQubits; ++q)
        if (std::find(targetQubits.begin(), targetQubits.end(), q) == targetQubits.end())
            nonTarget.push_back(q);

    // Control mask (|1> assumed)
    int controlMask = 0;
    for (int cq : controlQubits) controlMask |= (1 << cq);

    // Buffers
    hipDoubleComplex* d_block = nullptr;
    CHECK_CUDA(hipMalloc(&d_block, d * sizeof(hipDoubleComplex)));
    std::vector<int> h_offsets(d);
    int* d_offsets = nullptr;
    CHECK_CUDA(hipMalloc(&d_offsets, d * sizeof(int)));

    const int threads = 256;
    const int blocks  = (d + threads - 1) / threads;

    for (int blk = 0; blk < nBlocks; ++blk) {
        int nonTargetMask = 0;
        for (size_t i = 0; i < nonTarget.size(); ++i)
            if ((blk >> i) & 1) nonTargetMask |= (1 << nonTarget[i]);

        // Check controls satisfied
        if ((nonTargetMask & controlMask) != controlMask) continue;

        // Build offsets for this block
        for (int b = 0; b < d; ++b) {
            int targetMask = 0;
            for (int q = 0; q < k; ++q)
                if ((b >> q) & 1) targetMask |= (1 << targetQubits[q]);
            h_offsets[b] = nonTargetMask | targetMask;
        }
        CHECK_CUDA(hipMemcpy(d_offsets, h_offsets.data(), d * sizeof(int), hipMemcpyHostToDevice));

        // Gather subspace slice
        gather_kernel CUDA_KERNEL(blocks, threads) (d_block, d_state, d_offsets, d);
        CHECK_CUDA(hipDeviceSynchronize());

        // Apply Σ gamma[i] A^i to the d-dimensional block
        int rc = expiAv_chebyshev_gamma_cusparse_device(
            handle, d, nnz, order,
            d_csrRowPtr, d_csrColInd, d_csrVal,
            gamma,
            d_block);
        if (rc) { hipFree(d_offsets); hipFree(d_block); return rc; }

        // Scatter back
        scatter_kernel CUDA_KERNEL(blocks, threads) (d_state, d_block, d_offsets, d);
        CHECK_CUDA(hipDeviceSynchronize());
    }

    hipFree(d_offsets);
    hipFree(d_block);
    return 0;
}

// ==========================================================
// Controlled exp(iA) — build gamma from HOST CSR (copied from DEVICE),
// then call device version per-block
// ==========================================================
int applyControlledExpChebyshev_cusparse_host(
    hipsparseHandle_t handle,
    int nQubits,
    const int* d_csrRowPtr,                  // DEVICE CSR (dim d)
    const int* d_csrColInd,
    const hipDoubleComplex* d_csrVal,
    hipDoubleComplex* d_state,
    const std::vector<int>& targetQubits,
    const std::vector<int>& controlQubits,
    int nnz,
    int order,
    const double t /* = +1 for exp(-iA), -1 for exp(+iA) */)
{
    const int k = static_cast<int>(targetQubits.size());
    const int d = 1 << k;

    // Pull CSR back to HOST to compute gamma
    std::vector<int> h_r(d + 1), h_c(nnz);
    std::vector<hipDoubleComplex> h_v(nnz);
    CHECK_CUDA(hipMemcpy(h_r.data(), d_csrRowPtr, sizeof(int)*(d+1), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_c.data(), d_csrColInd, sizeof(int)*nnz,  hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_v.data(), d_csrVal,    sizeof(hipDoubleComplex)*nnz, hipMemcpyDeviceToHost));

    std::vector<hipDoubleComplex> gamma =
        chebyshev_exp_gamma_spectral_csr(
            d,
            std::span<const int>(h_r.data(), d+1),
            std::span<const int>(h_c.data(), nnz),
            std::span<const hipDoubleComplex>(h_v.data(), nnz),
            t, order, std::nullopt);

    // Apply on device block-wise with precomputed gamma
    return applyControlledExpChebyshev_cusparse_device(
        handle,
        nQubits,
        d_csrRowPtr, d_csrColInd, d_csrVal,
        gamma,
        d_state,
        targetQubits, controlQubits,
        nnz,
        order);
}
