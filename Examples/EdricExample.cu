
#include <complex>
#include <vector>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>
#include <cstdlib>
#include <cstdint>

#include <iostream>
#include <c++/11/bits/specfun.h>

#ifdef __HIPCC__
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#else
#define CUDA_KERNEL(...) 
#endif

#define nullify

#define CUDA_CHECK(err)                                                                                      \
    do                                                                                                       \
    {                                                                                                        \
        hipError_t err_ = (err);                                                                            \
        if (err_ != hipSuccess)                                                                             \
        {                                                                                                    \
            std::cerr << "CUDA error at line " << __LINE__ << ": " << hipGetErrorString(err_) << std::endl; \
            std::exit(EXIT_FAILURE);                                                                         \
        }                                                                                                    \
    } while (0)

#define CUSPARSE_CHECK(stat)                                                 \
    do                                                                       \
    {                                                                        \
        hipsparseStatus_t stat_ = (stat);                                     \
        if (stat_ != HIPSPARSE_STATUS_SUCCESS)                                \
        {                                                                    \
            std::cerr << "cuSPARSE error at line " << __LINE__ << std::endl; \
            std::exit(EXIT_FAILURE);                                         \
        }                                                                    \
    } while (0)

int calcGridSize(int64_t N, int blockSize)
{
    return (N + blockSize - 1) / blockSize;
}

double norm_squared(const std::vector<hipDoubleComplex> &v)
{
    double sum = 0.0;
    for (auto &elem : v)
    {
        double re = hipCreal(elem);
        double im = hipCimag(elem);
        sum += re * re + im * im;
    }
    return sum;
}

// Hypercube adjacency matrix in the CSR format - only generating rowOffsets and colIndices.
// For the values we fill an array of size numVertices with ones.
void generateHypercubeCSR(int n, std::vector<int64_t> &rowOffsets, std::vector<int64_t> &colIndices)
{
    int64_t numVertices = (int64_t)1 << n;

    rowOffsets.resize((size_t)numVertices + 1);
    colIndices.reserve((size_t)n * (size_t)numVertices); // each vertex has n neighbors

    int64_t edgeCount = 0;
    for (int64_t i = 0; i < numVertices; ++i)
    {
        rowOffsets[(size_t)i] = edgeCount;
        for (int bit = 0; bit < n; ++bit)
        {
            int64_t neighbor = i ^ ((int64_t)1 << bit);
            colIndices.push_back(neighbor);
            edgeCount++;
        }
    }
    rowOffsets[(size_t)numVertices] = edgeCount;
}

__global__ void scale_kernel(hipDoubleComplex *x, int64_t N, double alpha_real, double alpha_imag)
{
    hipDoubleComplex alpha = make_hipDoubleComplex(alpha_real, alpha_imag);
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        x[idx] = hipCmul(alpha, x[idx]);
    }
}

__global__ void axpy_kernel(hipDoubleComplex *y, const hipDoubleComplex *x, int64_t N, double alpha_real, double alpha_imag)
{
    hipDoubleComplex alpha = make_hipDoubleComplex(alpha_real, alpha_imag);
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        y[idx] = hipCadd(y[idx], hipCmul(alpha, x[idx]));
    }
}

void scale_vector(hipDoubleComplex *x, int64_t N, std::complex<double> alpha)
{
    int blockSize = 256;
    int64_t gridSize = calcGridSize(N, blockSize);
    scale_kernel CUDA_KERNEL((unsigned int)gridSize, blockSize) (x, N, alpha.real(), alpha.imag());
    CUDA_CHECK(hipDeviceSynchronize());
}

void axpy_vector(hipDoubleComplex *y, const hipDoubleComplex *x, int64_t N, std::complex<double> alpha)
{
    int blockSize = 256;
    int64_t gridSize = calcGridSize(N, blockSize);
    axpy_kernel CUDA_KERNEL((unsigned int)gridSize, blockSize) (y, x, N, alpha.real(), alpha.imag());
    CUDA_CHECK(hipDeviceSynchronize());
}

int main()
{
    int n = 6; // number of qubits
    int64_t numVertices = (int64_t)1 << n;

    std::vector<int64_t> rowOffsets, colIndices; // using 64-bit indexing for n > 31
    generateHypercubeCSR(n, rowOffsets, colIndices);
    int64_t nnz = (int64_t)colIndices.size();

    double M = double(n);
    double t = 1.0;

    // CSR values array (all ones)
    std::vector<hipDoubleComplex> h_csrVal((size_t)nnz);
    std::fill(h_csrVal.begin(), h_csrVal.end(), make_hipDoubleComplex(1.0, 0.0));

    // Initial state
    std::vector<hipDoubleComplex> h_psi((size_t)numVertices);
    std::fill(h_psi.begin(), h_psi.end(), make_hipDoubleComplex(0.0, 0.0));
    h_psi[0].x = 1.0;

    // Allocate and transfer to device memory
    hipDoubleComplex *d_csrVal, *d_psi, *d_w0, *d_w1, *d_result;
    int64_t *d_csrRowPtr, *d_csrColInd;

    CUDA_CHECK(hipMalloc((void **)&d_csrVal, sizeof(hipDoubleComplex) * nnz));
    CUDA_CHECK(hipMalloc((void **)&d_csrRowPtr, sizeof(int64_t) * (numVertices + 1)));
    CUDA_CHECK(hipMalloc((void **)&d_csrColInd, sizeof(int64_t) * nnz));

    CUDA_CHECK(hipMemcpy(d_csrVal, h_csrVal.data(), sizeof(hipDoubleComplex) * nnz, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_csrRowPtr, rowOffsets.data(), sizeof(int64_t) * (numVertices + 1), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_csrColInd, colIndices.data(), sizeof(int64_t) * nnz, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void **)&d_psi, sizeof(hipDoubleComplex) * numVertices));
    CUDA_CHECK(hipMalloc((void **)&d_w0, sizeof(hipDoubleComplex) * numVertices));
    CUDA_CHECK(hipMalloc((void **)&d_w1, sizeof(hipDoubleComplex) * numVertices));
    CUDA_CHECK(hipMalloc((void **)&d_result, sizeof(hipDoubleComplex) * numVertices));

    CUDA_CHECK(hipMemcpy(d_psi, h_psi.data(), sizeof(hipDoubleComplex) * numVertices, hipMemcpyHostToDevice));

    // Create cuSPARSE handle and descriptors
    hipsparseHandle_t handle;
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    hipsparseSpMatDescr_t matDescr;
    hipsparseDnVecDescr_t vecX, vecY;

    // Using 64-bit indexing in cuSPARSE
    CUSPARSE_CHECK(hipsparseCreateCsr(&matDescr,
                                     (int64_t)numVertices, (int64_t)numVertices, nnz,
                                     d_csrRowPtr, d_csrColInd, d_csrVal,
                                     HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F));

    CUSPARSE_CHECK(hipsparseCreateDnVec(&vecX, (int64_t)numVertices, d_w0, HIP_C_64F));
    CUSPARSE_CHECK(hipsparseCreateDnVec(&vecY, (int64_t)numVertices, d_w1, HIP_C_64F));

    // Buffer for SpMV
    size_t bufferSize = 0;
    void *dBuffer = nullptr;
    hipDoubleComplex alpha = {1.0, 0.0}, beta = {0.0, 0.0};
    CUSPARSE_CHECK(hipsparseSpMV_bufferSize(handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha,
                                           matDescr,
                                           vecX,
                                           &beta,
                                           vecY,
                                           HIP_C_64F,
                                           HIPSPARSE_SPMV_ALG_DEFAULT,
                                           &bufferSize));
    CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    double epsilon = 1e-15; // target precision
    double z = t * M;
    std::complex<double> minus_i(0.0, -1.0);

    // Determine the polynomial degree m
    std::vector<std::complex<double>> ck;
    int k = 0;

    double J0 = std::cyl_bessel_j(0, z);
    ck.push_back(J0); // c0 = J0(z)

    if (std::abs(J0) >= epsilon)
    {
        while (true)
        {
            k++;
            double Jk = std::cyl_bessel_j(k, z);
            std::complex<double> ck_val = 2.0 * std::pow(minus_i, k) * Jk;
            ck.push_back(ck_val);

            if (std::abs(Jk) < epsilon)
            {
                break;
            }
        }
    }

    int m = k;
    std::cout << "Selected polynomial degree m = " << m << std::endl;

    // w0 = v
    CUDA_CHECK(hipMemcpy(d_w0, d_psi, sizeof(hipDoubleComplex) * numVertices, hipMemcpyDeviceToDevice));

    // Scale to get H_tilde = H/M
    for (int64_t i = 0; i < nnz; i++)
    {
        h_csrVal[(size_t)i] = make_hipDoubleComplex(1.0 / M, 0.0);
    }
    CUDA_CHECK(hipMemcpy(d_csrVal, h_csrVal.data(), sizeof(hipDoubleComplex) * nnz, hipMemcpyHostToDevice));

    // w1 = H_tilde * w0
    CUSPARSE_CHECK(hipsparseDnVecSetValues(vecX, d_w0));
    CUSPARSE_CHECK(hipsparseDnVecSetValues(vecY, d_w1));
    hipsparseSpMV(handle,
                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                 &alpha,
                 matDescr,
                 vecX,
                 &beta,
                 vecY,
                 HIP_C_64F,
                 HIPSPARSE_SPMV_ALG_DEFAULT,
                 dBuffer);

    // result = c0*w0 + c1*w1
    CUDA_CHECK(hipMemcpy(d_result, d_w0, sizeof(hipDoubleComplex) * numVertices, hipMemcpyDeviceToDevice));
    scale_vector(d_result, numVertices, ck[0]);
    axpy_vector(d_result, d_w1, numVertices, ck[1]);

    hipDoubleComplex *d_wtemp;
    CUDA_CHECK(hipMalloc((void **)&d_wtemp, sizeof(hipDoubleComplex) * numVertices));

    for (int i_iter = 2; i_iter <= m; i_iter++)
    {
        CUDA_CHECK(hipMemcpy(d_wtemp, d_w0, sizeof(hipDoubleComplex) * numVertices, hipMemcpyDeviceToDevice));

        // w0 = w1
        CUDA_CHECK(hipMemcpy(d_w0, d_w1, sizeof(hipDoubleComplex) * numVertices, hipMemcpyDeviceToDevice));

        // w1 = H_tilde*w0
        CUSPARSE_CHECK(hipsparseDnVecSetValues(vecX, d_w0));
        CUSPARSE_CHECK(hipsparseDnVecSetValues(vecY, d_w1));

        hipsparseSpMV(handle,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &alpha,
                     matDescr,
                     vecX,
                     &beta,
                     vecY,
                     HIP_C_64F,
                     HIPSPARSE_SPMV_ALG_DEFAULT,
                     dBuffer);

        // w1 = 2*w1 - wtemp
        scale_vector(d_w1, numVertices, std::complex<double>(2.0, 0.0));
        axpy_vector(d_w1, d_wtemp, numVertices, std::complex<double>(-1.0, 0.0));

        // result += c_i*w1
        axpy_vector(d_result, d_w1, numVertices, ck[i_iter]);
    }

    CUDA_CHECK(hipFree(d_wtemp));

    // Copy result back to host - can avoid this by using managed memory...
    std::vector<hipDoubleComplex> h_result((size_t)numVertices);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, sizeof(hipDoubleComplex) * numVertices, hipMemcpyDeviceToHost));

    std::cout << "Result (first few elements):\n";
    for (int64_t i = 0; i < std::min<int64_t>(numVertices, 5); i++)
    {
        std::complex<double> val(hipCreal(h_result[(size_t)i]), hipCimag(h_result[(size_t)i]));
        std::cout << val << "\n";
    }

    double norm = norm_squared(h_result);
    std::cout << norm << std::endl;

    CUSPARSE_CHECK(hipsparseDestroyDnVec(vecX));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(vecY));
    CUSPARSE_CHECK(hipsparseDestroySpMat(matDescr));
    CUSPARSE_CHECK(hipsparseDestroy(handle));

    CUDA_CHECK(hipFree(d_csrVal));
    CUDA_CHECK(hipFree(d_csrRowPtr));
    CUDA_CHECK(hipFree(d_csrColInd));
    CUDA_CHECK(hipFree(d_psi));
    CUDA_CHECK(hipFree(d_w0));
    CUDA_CHECK(hipFree(d_w1));
    CUDA_CHECK(hipFree(d_result));
    CUDA_CHECK(hipFree(dBuffer));

    return 0;
}