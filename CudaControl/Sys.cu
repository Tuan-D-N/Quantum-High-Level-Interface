
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../functionality/WriteAdjMat.hpp"
#include <hip/hip_complex.h>
#include <iostream>
#include "Sys.hpp"

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    }

template <typename T>
void printDeviceArray(T *d_array, T size)
{
    T *h_array = new T[size];
    hipMemcpy(h_array, d_array, size * sizeof(T), hipMemcpyDeviceToHost);
    for (T i = 0; i < size; ++i)
        std::cout << h_array[i] << " ";
    std::cout << std::endl;
    delete[] h_array;
}

void printDeviceArray(hipDoubleComplex *d_array, int size)
{
    hipDoubleComplex *h_array = new hipDoubleComplex[size];
    hipMemcpy(h_array, d_array, size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i)
        std::cout << "(" << h_array[i].x << ", " << h_array[i].y << ") ";
    std::cout << std::endl;

    delete[] h_array;
}

int runSys()
{
    // Host problem definition
    int evenqubits = 4;
    int A_num_rows = 1 << evenqubits;
    int A_num_cols = 1 << evenqubits;
    int A_max_nnz = 4 * A_num_rows;

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    //--------------------------------------------------------------------------
    // Device memory management
    int *dA_csrOffsets, *dA_columns;
    hipDoubleComplex *dA_values;
    CHECK_CUDA(hipMallocManaged((void **)&dA_csrOffsets,
                                 (A_num_rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_columns, A_max_nnz * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_values, A_max_nnz * sizeof(hipDoubleComplex)))

    int postIndexSize, postOffsetSize, postValueSize;

    // Unified Memory Cuda Write
    writeMatAMiniCSC(dA_csrOffsets, dA_columns, dA_values, evenqubits, postOffsetSize, postIndexSize, postValueSize);

    // Vector
    hipDoubleComplex *rThetaVector;
    hipDoubleComplex *xyVector;
    CHECK_CUDA(hipMallocManaged((void **)&rThetaVector, A_num_cols * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMallocManaged((void **)&xyVector, A_num_cols * sizeof(hipDoubleComplex)));

    for (int i = 0; i < A_num_cols; ++i)
    {
        xyVector[0] = make_hipDoubleComplex(0, 0);
        rThetaVector[i] = make_hipDoubleComplex(0, 0);
    }
    rThetaVector[0] = make_hipDoubleComplex(1, 0);

    //--------------------------------------------------------------------------
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, postValueSize,
                                     dA_csrOffsets, dA_columns, dA_values,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F))

    hipsparseDnVecDescr_t  vectorIn;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorIn, A_num_rows, rThetaVector, HIP_C_64F));

    hipsparseDnVecDescr_t vectorOut;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorOut, A_num_rows, xyVector, HIP_C_64F));

    //---------------------------------------------------------------------------

    // Workspace buffer
    void *dBuffer = nullptr;
    size_t bufferSize = 0;
    float tmp_result;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                            &alpha, matA, vectorIn, &beta, vectorOut,
                            HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // Perform the SpMV operation
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                 &alpha, matA, vectorIn, &beta, vectorOut,
                 HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

    printDeviceArray(dA_csrOffsets, postOffsetSize);
    printDeviceArray(dA_columns, postIndexSize);
    printDeviceArray(dA_values, postValueSize);

    std::cout << alpha.x << "," << alpha.y << "\n";
    std::cout <<  beta.x << "," << beta.y << "\n";
    
    printDeviceArray(rThetaVector, A_num_cols);
    printDeviceArray(xyVector, A_num_cols);

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vectorIn);
    hipsparseDestroyDnVec(vectorOut);


    CHECK_CUDA(hipFree(dA_csrOffsets))
    CHECK_CUDA(hipFree(dA_columns))
    CHECK_CUDA(hipFree(dA_values))
    CHECK_CUDA(hipFree(xyVector))
    CHECK_CUDA(hipFree(rThetaVector))
    return EXIT_SUCCESS;
}