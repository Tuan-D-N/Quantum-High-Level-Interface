
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../functionality/WriteAdjMat.hpp"
#include "../functionality/ReadCsv.hpp"
#include "../functionality/Utilities.hpp"
#include "../functionality/fftShift.hpp"
#include "../CuQuantumControl/QftStateVec.hpp"
#include <hip/hip_complex.h>
#include <iostream>
#include "Sys.hpp"
#include <string>
#include <cassert>

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    }

template <typename T>
void printDeviceArray(T *d_array, T size)
{
    T *h_array = new T[size];
    hipMemcpy(h_array, d_array, size * sizeof(T), hipMemcpyDeviceToHost);
    for (T i = 0; i < size; ++i)
        std::cout << h_array[i] << " ";
    std::cout << std::endl;
    delete[] h_array;
}

void printDeviceArray(hipDoubleComplex *d_array, int size)
{
    hipDoubleComplex *h_array = new hipDoubleComplex[size];
    hipMemcpy(h_array, d_array, size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i)
        std::cout << "(" << h_array[i].x << ", " << h_array[i].y << ") ";
    std::cout << std::endl;

    delete[] h_array;
}

void getData(hipDoubleComplex *rThetaVector, const int evenqubits, const std::string fileName)
{
    int lengthSize = 1 << (evenqubits / 2);
    std::vector<std::vector<float>> image = readCSV<float>(fileName);
    assert(image.size() == lengthSize);

    for (int i = 0; i < lengthSize; ++i)
    {
        assert(image[i].size() == lengthSize);
        for (int j = 0; j < lengthSize; ++j)
        {
            rThetaVector[i * lengthSize + j] = {image[i][j], 0};
        }
    }
}

void applyQFTHorizontally(hipDoubleComplex *vector, const int num_columns, const int num_rows, const int num_qubit_per_row)
{
    for (int i = 0; i < num_rows; ++i)
    {
        ApplyQFTOnStateVector(&vector[i * num_columns], num_qubit_per_row);
    }
}

void applyQFTVertically(hipDoubleComplex *vector, hipDoubleComplex *workSpace, const int num_columns, const int num_rows, const int num_qubit_per_row)
{
    for (int i = 0; i < num_columns; ++i)
    {
        for (int j = 0; j < num_rows; ++j)
        {
            workSpace[j] = vector[i * num_rows + j];
        }

        ApplyQFTOnStateVector(workSpace, num_qubit_per_row);

        for (int j = 0; j < num_rows; ++j)
        {
            vector[i * num_rows + j] = workSpace[j];
        }
    }
}

int runSys()
{
    // Host problem definition
    int evenqubits = 4;
    int halfOfQubits = evenqubits / 2;
    int svSize = 1 << evenqubits;
    int img_num_rows = 1 << (halfOfQubits);
    int img_num_columns = 1 << (halfOfQubits);
    int A_num_rows = 1 << evenqubits;
    int A_num_cols = 1 << evenqubits;
    int A_max_nnz = 4 * A_num_rows;

    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    //--------------------------------------------------------------------------
    // Device memory management
    int *dA_csrOffsets, *dA_columns;
    hipDoubleComplex *dA_values;
    CHECK_CUDA(hipMallocManaged((void **)&dA_csrOffsets,
                                 (A_num_rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_columns, A_max_nnz * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_values, A_max_nnz * sizeof(hipDoubleComplex)))

    int postIndexSize, postOffsetSize, postValueSize;

    // Unified Memory Cuda Write
    writeMatAMiniCSC(dA_csrOffsets, dA_columns, dA_values, evenqubits, postOffsetSize, postIndexSize, postValueSize);

    // Vector
    hipDoubleComplex *rThetaVector; // theta slow, r fast
    hipDoubleComplex *xyVector;
    CHECK_CUDA(hipMallocManaged((void **)&rThetaVector, A_num_cols * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMallocManaged((void **)&xyVector, A_num_cols * sizeof(hipDoubleComplex)));

    for (int i = 0; i < A_num_cols; ++i)
    {
        xyVector[0] = make_hipDoubleComplex(0, 0);
        rThetaVector[i] = make_hipDoubleComplex(0, 0);
    }
    getData(rThetaVector, evenqubits, "../imageFile.csv");

    printDeviceArray(rThetaVector, svSize);
    fftshift2D(rThetaVector, img_num_rows, img_num_columns);
    printDeviceArray(rThetaVector, svSize);

    hipDoubleComplex *qftWorkSpace;
    CHECK_CUDA(hipMallocManaged((void **)&qftWorkSpace, img_num_rows * sizeof(hipDoubleComplex)));

    for (int i = 0; i < img_num_rows; ++i)
    {
        ApplyQFTOnStateVector(&rThetaVector[i * img_num_columns], halfOfQubits);
    }
    printDeviceArray(rThetaVector, svSize);
    
    CHECK_CUDA(hipFree(qftWorkSpace))
    //--------------------------------------------------------------------------
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, postValueSize,
                                     dA_csrOffsets, dA_columns, dA_values,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F))

    hipsparseDnVecDescr_t vectorIn;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorIn, A_num_rows, rThetaVector, HIP_C_64F));

    hipsparseDnVecDescr_t vectorOut;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vectorOut, A_num_rows, xyVector, HIP_C_64F));

    //---------------------------------------------------------------------------

    // Workspace buffer
    void *dBuffer = nullptr;
    size_t bufferSize = 0;
    float tmp_result;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                                           &alpha, matA, vectorIn, &beta, vectorOut,
                                           HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // Perform the SpMV operation
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE,
                                &alpha, matA, vectorIn, &beta, vectorOut,
                                HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

    printDeviceArray(dA_csrOffsets, postOffsetSize);
    printDeviceArray(dA_columns, postIndexSize);
    printDeviceArray(dA_values, postValueSize);

    std::cout << alpha.x << "," << alpha.y << "\n";
    std::cout << beta.x << "," << beta.y << "\n";

    printDeviceArray(rThetaVector, A_num_cols);
    printDeviceArray(xyVector, A_num_cols);

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vectorIn);
    hipsparseDestroyDnVec(vectorOut);


    CHECK_CUDA(hipFree(dA_csrOffsets))
    CHECK_CUDA(hipFree(dA_columns))
    CHECK_CUDA(hipFree(dA_values))
    CHECK_CUDA(hipFree(xyVector))
    CHECK_CUDA(hipFree(rThetaVector))
    return EXIT_SUCCESS;
}