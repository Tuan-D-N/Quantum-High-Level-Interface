
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../functionality/WriteAdjMat.hpp"
#include <hip/hip_complex.h>
#include <iostream>
#include "Sys.hpp"

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    }

template<typename T>
void printDeviceArray(T* d_array, T size) {
    T* h_array = new T[size];
    hipMemcpy(h_array, d_array, size * sizeof(T), hipMemcpyDeviceToHost);
    for (T i = 0; i < size; ++i)
        std::cout << h_array[i] << " ";
    std::cout << std::endl;
    delete[] h_array;
}

void printDeviceArray(hipDoubleComplex* d_array, int size) {
    hipDoubleComplex* h_array = new hipDoubleComplex[size];
    hipMemcpy(h_array, d_array, size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < size; ++i)
        std::cout << "(" << h_array[i].x << ", " << h_array[i].y << ") ";
    std::cout << std::endl;
    
    delete[] h_array;
}

int runSys()
{
    // Host problem definition
    int evenqubits = 4;
    int A_num_rows = 1 << evenqubits;
    int A_num_cols = 1 << evenqubits;
    int A_max_nnz = 4*A_num_rows;

    float alpha = 1.0f;
    float beta = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int *dA_csrOffsets, *dA_columns;
    hipDoubleComplex *dA_values;
    CHECK_CUDA(hipMallocManaged((void **)&dA_csrOffsets,
                                 (A_num_rows + 1) * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_columns, A_max_nnz * sizeof(int)))
    CHECK_CUDA(hipMallocManaged((void **)&dA_values, A_max_nnz * sizeof(hipDoubleComplex)))

    int postIndexSize, postOffsetSize, postValueSize;

    writeMatAMiniCSC(dA_csrOffsets, dA_columns, dA_values, evenqubits, postOffsetSize, postIndexSize, postValueSize);
    //--------------------------------------------------------------------------

    printDeviceArray(dA_csrOffsets, postOffsetSize);
    printDeviceArray(dA_columns, postOffsetSize);
    printDeviceArray(dA_values, postOffsetSize);
    

    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    return EXIT_SUCCESS;
}