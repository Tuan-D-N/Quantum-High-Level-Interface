
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../functionality/WriteAdjMat.hpp"
#include "../functionality/ReadCsv.hpp"
#include "../functionality/Utilities.hpp"
#include "../functionality/fftShift.hpp"
#include "../functionality/Transpose.hpp"
#include "../CuQuantumControl/QftStateVec.hpp"
#include "../CudaControl/Helper.hpp"
#include "../CuSparseControl/ApplyMatrixA.hpp"
#include <hip/hip_complex.h>
#include <iostream>
#include "ApplyIRadon.hpp"
#include <string>
#include <cassert>
#include <span>
#include <iostream>

void printSpan(std::span<const int> span)
{
    for (int value : span)
    {
        std::cout << value << " ";
    }
    std::cout << "\n";
}

void printSpan(int span)
{
    std::cout << "\n";
}

int main()
{
    printSpan(std::array<int,5>{1, 2, 3, 4, 5}); // Compiler deduces the size as 5

    return 0;
}