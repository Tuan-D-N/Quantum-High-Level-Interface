#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "../CudaControl/Helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "../CuQuantumControl/ApplyGates.hpp"
#include "../CuQuantumControl/ApplySampler.hpp"
#include "../CuQuantumControl/QftStateVec.hpp"
#include "../CuQuantumControl/Utilities.hpp"
#include "../functionality/fftShift.hpp"
#include "../functionality/ClockTimer.hpp"
#include "../functionality/RangeCompileTime.hpp"
#include "../functionality/Utilities.hpp"
#include "../functionality/randomArray.hpp"
#include <cstring>

int grover(const int nIndexBits)
{
    using cuType = hipComplex;
    const auto cuStateVecComputeType = CUSTATEVEC_COMPUTE_32F;
    const auto cuStateVecCudaDataType = HIP_C_32F;
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;
    const int nShots = 100;
    {
        auto timer = Timer("Grover Cuquantum C++ qubits = " + std::to_string(nIndexBits));

        // Grover ----------------------------------------------------------------------------------------
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Make the statevector -------------------------------------------------------------------------------
        cuType *d_sv;
        CHECK_CUDA(hipMalloc((void **)&d_sv, nSvSize * sizeof(cuType)));
        // initialize the state vector
        CHECK_CUSTATEVECTOR(custatevecInitializeStateVector(
            handle, d_sv, cuStateVecCudaDataType, nIndexBits, CUSTATEVEC_STATE_VECTOR_TYPE_ZERO));
        // Make the statevector -------------------------------------------------------------------------------

        // Algo ------------------------------------------------------------
        std::vector<int> allQubit = rangeVec(0, nIndexBits);
        std::vector<int> allQubitExceptLast = rangeVec(0, nIndexBits - 1);

        CHECK_BROAD_ERROR(applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

        for (int i = 0; i < 10; ++i)
        {
            // Mark
            int markTarget = nIndexBits - 1; // lastQubit
            CHECK_BROAD_ERROR(applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

            // Diffusion
            CHECK_BROAD_ERROR(applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        std::vector<custatevecIndex_t> outBitString;
        CHECK_BROAD_ERROR(sampleSV(handle, nIndexBits, allQubit, d_sv, outBitString, nShots, extraWorkspace, extraWorkspaceSizeInBytes));

        // Algo ------------------------------------------------------------
        CHECK_BROAD_ERROR(custatevecDestroy(handle));
        if (extraWorkspace != nullptr)
            CHECK_CUDA(hipFree(extraWorkspace));

        // Grover ----------------------------------------------------------------------------------------
        CHECK_CUDA(hipFree(d_sv));
    }

    return hipSuccess;
}

int grover2(const int nIndexBits)
{
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;
    const int nShots = 100;
    {
        auto timer = Timer("Grover Cuquantum C++ qubits = " + std::to_string(nIndexBits));

        // Make the statevector -------------------------------------------------------------------------------
        hipDoubleComplex *d_sv;
        CHECK_CUDA(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));
        d_sv[0] = {1, 0};
        for (int i = 1; i < nSvSize; ++i)
        {
            d_sv[i] = {0, 0};
        }
        // Make the statevector -------------------------------------------------------------------------------

        // Grover ----------------------------------------------------------------------------------------
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Algo ------------------------------------------------------------
        std::vector<int> allQubit = rangeVec(0, nIndexBits);
        std::vector<int> allQubitExceptLast = rangeVec(0, nIndexBits - 1);

        (applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

        for (int i = 0; i < 10; ++i)
        {
            // Mark
            int markTarget = nIndexBits - 1; // lastQubit
            (applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

            // Diffusion
            (applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        std::vector<custatevecIndex_t> outBitString;
        (sampleSV(handle, nIndexBits, allQubit, d_sv, outBitString, nShots, extraWorkspace, extraWorkspaceSizeInBytes));

        // Algo ------------------------------------------------------------
        CHECK_CUSTATEVECTOR(custatevecDestroy(handle));
        if (extraWorkspace != nullptr)
            CHECK_CUDA(hipFree(extraWorkspace));

        // Grover ----------------------------------------------------------------------------------------
        CHECK_CUDA(hipFree(d_sv));
    }

    return hipSuccess;
}
