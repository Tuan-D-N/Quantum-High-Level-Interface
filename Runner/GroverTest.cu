#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <bitset>
#include "../CudaControl/Helper.hpp" // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include "../CuQuantumControl/ApplyGates.hpp"
#include "../CuQuantumControl/ApplySampler.hpp"
#include "../CuQuantumControl/QftStateVec.hpp"
#include "../CuQuantumControl/Utilities.hpp"
#include "../functionality/fftShift.hpp"
#include "../functionality/ClockTimer.hpp"
#include "../functionality/RangeCompileTime.hpp"
#include "../functionality/Utilities.hpp"
#include "../functionality/randomArray.hpp"
#include <cstring>

int grover(const int nIndexBits)
{
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;
    const int nShots = 100;
    {
        auto timer = Timer("Grover Cuquantum C++ qubits = " + std::to_string(nIndexBits));

        // Make the statevector -------------------------------------------------------------------------------
        hipDoubleComplex *d_sv;
        CHECK_CUDA(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));
        d_sv[0] = {1, 0};
        for (int i = 1; i < nSvSize; ++i)
        {
            d_sv[i] = {0, 0};
        }
        // Make the statevector -------------------------------------------------------------------------------

        // Grover ----------------------------------------------------------------------------------------
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Algo ------------------------------------------------------------
        std::vector<int> allQubit = rangeVec(0, nIndexBits);
        std::vector<int> allQubitExceptLast = rangeVec(0, nIndexBits - 1);

        CHECK_BROAD_ERROR(applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

        for (int i = 0; i < 10; ++i)
        {
            // Mark
            int markTarget = nIndexBits - 1; // lastQubit
            CHECK_BROAD_ERROR(applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

            // Diffusion
            CHECK_BROAD_ERROR(applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            CHECK_BROAD_ERROR(applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        std::vector<custatevecIndex_t> outBitString;
        CHECK_BROAD_ERROR(sampleSV(handle, nIndexBits, allQubit, d_sv, outBitString, nShots, extraWorkspace, extraWorkspaceSizeInBytes));

        // Algo ------------------------------------------------------------
        CHECK_BROAD_ERROR(custatevecDestroy(handle));
        if (extraWorkspace != nullptr)
            CHECK_CUDA(hipFree(extraWorkspace));

        // Grover ----------------------------------------------------------------------------------------
        CHECK_CUDA(hipFree(d_sv));
    }

    return hipSuccess;
}

int grover2(const int nIndexBits)
{
    const int nSvSize = (1 << nIndexBits);
    const int adjoint = 0;
    const int nShots = 100;
    {
        auto timer = Timer("Grover Cuquantum C++ qubits = " + std::to_string(nIndexBits));

        // Make the statevector -------------------------------------------------------------------------------
        hipDoubleComplex *d_sv;
        CHECK_CUDA(hipMallocManaged((void **)&d_sv, nSvSize * sizeof(hipDoubleComplex)));
        d_sv[0] = {1, 0};
        for (int i = 1; i < nSvSize; ++i)
        {
            d_sv[i] = {0, 0};
        }
        // Make the statevector -------------------------------------------------------------------------------

        // Grover ----------------------------------------------------------------------------------------
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Algo ------------------------------------------------------------
        std::vector<int> allQubit = rangeVec(0, nIndexBits);
        std::vector<int> allQubitExceptLast = rangeVec(0, nIndexBits - 1);

        (applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

        for (int i = 0; i < 10; ++i)
        {
            // Mark
            int markTarget = nIndexBits - 1; // lastQubit
            (applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));

            // Diffusion
            (applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyZ(handle, nIndexBits, adjoint, markTarget, allQubitExceptLast, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyX(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
            (applyH(handle, nIndexBits, adjoint, allQubit, d_sv, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        std::vector<custatevecIndex_t> outBitString;
        (sampleSV(handle, nIndexBits, allQubit, d_sv, outBitString, nShots, extraWorkspace, extraWorkspaceSizeInBytes));

        // Algo ------------------------------------------------------------
        CHECK_CUSTATEVECTOR(custatevecDestroy(handle));
        if (extraWorkspace != nullptr)
            CHECK_CUDA(hipFree(extraWorkspace));

        // Grover ----------------------------------------------------------------------------------------
        CHECK_CUDA(hipFree(d_sv));
    }

    return hipSuccess;
}

template <int nIndexBits>
int grover3()
{
    const int svSize = (1 << nIndexBits);

    const int nShots = 100;
    const int nMaxShots = nShots;
    const int bitOrdering[svSize];
    for (int i = 0; i < svSize; ++i)
    {
        bitOrdering[i] = i;
    }
    const int bitStringLen = svSize;
    custatevecIndex_t bitStrings[nShots];
    double randnums[nShots] = {};
    generateRandomArray(randnums, nShots);

    hipDoubleComplex xMat[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};
    hipDoubleComplex zMat[] = {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {-1.0, 0.0}};
    hipDoubleComplex hMat[] = {{INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {-INV_SQRT2, 0.0}};

    hipDoubleComplex *d_sv;
    CHECK_CUDA(hipMallocManaged((void **)&d_sv, svSize * sizeof(hipDoubleComplex)));

    //----------------------------------------------------------------------------------------------

    {
        auto timer = Timer("Grover Cuquantum C++ qubits = " + std::to_string(nIndexBits));

        int controlsAll[nIndexBits];
        int controlsAllExceptLast[nIndexBits - 1];
        int markTargets[] = {nIndexBits - 1};
        for (int i = 0; i < nIndexBits - 1; ++i)
        {
            controlsAll[i] = i;
            controlsAllExceptLast[i] = i;
        }
        controlsAll[nIndexBits - 1] = nIndexBits - 1;

        // custatevec handle initialization
        custatevecSamplerDescriptor_t sampler;
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Init to zero state
        d_sv[0] = {1, 0};
        for (int i = 1; i < svSize; ++i)
        {
            d_sv[i] = {0, 0};
        }
        // H to all qubits
        for (int i = 0; i < nIndexBits; ++i)
        {
            int targets[] = {i};
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, HIP_C_64F, nIndexBits, hMat, HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                0, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        // H to all qubits

        for (int i = 0; i < 10; ++i)
        {
            // mark
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, HIP_C_64F, nIndexBits, zMat, HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast, nullptr,
                nIndexBits - 1, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
            // Diffusion
            // H->all, X->all, cz->allexceptLast mark, x->all, H->all
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, HIP_C_64F, nIndexBits, hMat, HIP_C_64F,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, HIP_C_64F, nIndexBits, xMat, HIP_C_64F,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, HIP_C_64F, nIndexBits, zMat, HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast, nullptr,
                nIndexBits - 1, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, HIP_C_64F, nIndexBits, xMat, HIP_C_64F,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, HIP_C_64F, nIndexBits, hMat, HIP_C_64F,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes));
            }
        }

        // create sampler and check the size of external workspace
        CHECK_CUSTATEVECTOR(custatevecSamplerCreate(
            handle, d_sv, HIP_C_64F, nIndexBits, &sampler, nMaxShots,
            &extraWorkspaceSizeInBytes));

        // allocate external workspace if necessary
        if (extraWorkspaceSizeInBytes > 0)
            CHECK_CUDA(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

        // sample preprocess
        CHECK_CUSTATEVECTOR(custatevecSamplerPreprocess(
            handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes));

        // sample bit strings
        CHECK_CUSTATEVECTOR(custatevecSamplerSample(
            handle, sampler, bitStrings, bitOrdering, bitStringLen, randnums, nShots,
            CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

        // destroy descriptor and handle
        CHECK_CUSTATEVECTOR(custatevecSamplerDestroy(sampler));

        //  destroy handle
        CHECK_CUSTATEVECTOR(custatevecDestroy(handle));
        if (extraWorkspaceSizeInBytes)
            CHECK_CUDA(hipFree(extraWorkspace));
    }
    //----------------------------------------------------------------------------------------------

    printDeviceArray(d_sv, svSize);
    CHECK_CUDA(hipFree(d_sv));

    return EXIT_SUCCESS;
}