#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <string>
#include <custatevec.h>       // custatevecInitializeStateVector
#include <hip/hip_complex.h>
#include <random>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#define INV_SQRT2 (0.7071067811865475) // Approximation of 1/sqrt(2)

// Macro to check CUDA API errors
#define CHECK_CUDA(func)                                                          \
    {                                                                             \
        hipError_t status = (func);                                              \
        if (status != hipSuccess)                                                \
        {                                                                         \
            printf("CUDA API failed at line %d in file %s with error: %s (%d)\n", \
                   __LINE__, __FILE__, hipGetErrorString(status), status);       \
            return EXIT_FAILURE;                                                  \
        }                                                                         \
    }

// Macro to check cuSPARSE API errors
#define CHECK_CUSPARSE(func)                                                     \
    {                                                                            \
        hipsparseStatus_t status = (func);                                        \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                   \
        {                                                                        \
            printf("CUSPARSE API failed at line %d in file %s with error: %d\n", \
                   __LINE__, __FILE__, status);                                  \
            return EXIT_FAILURE;                                                 \
        }                                                                        \
    }

// Macro to check cuSPARSE API errors
#define CHECK_CUSTATEVECTOR(func)                                                \
    {                                                                            \
        custatevecStatus_t status = (func);                                      \
        if (status != CUSTATEVEC_STATUS_SUCCESS)                                 \
        {                                                                        \
            printf("CUSTATEVECTOR API failed at line %d in file %s with error: %d\n", \
                   __LINE__, __FILE__, status);                                  \
            return EXIT_FAILURE;                                                 \
        }                                                                        \
    }

#define CHECK_BROAD_ERROR(integer)                                    \
    {                                                                 \
        if (integer != 0)                                             \
        {                                                             \
            printf("Broad CUDA ERROR failed at line %d in file %s\n", \
                   __LINE__, __FILE__);                               \
            return EXIT_FAILURE;                                      \
        }                                                             \
    }


void generateRandomArray(double* arr, std::size_t size) {
    std::random_device rd; // Seed
    std::mt19937 gen(rd()); // Mersenne Twister RNG
    std::uniform_real_distribution<double> dist(0.0f, 1.0f); // Range [0, 1)

    for (std::size_t i = 0; i < size; ++i) {
        arr[i] = dist(gen);
    }
}

int main()
{
    const int nIndexBits = 30;
    const auto cuStateVecComputeType = CUSTATEVEC_COMPUTE_64F;
    const auto cuStateVecCudaDataType = HIP_C_64F;
    using cuType = hipDoubleComplex;
    constexpr int svSize = (1 << nIndexBits);

    const int nShots = 100;
    const int nMaxShots = nShots;
    int bitOrdering[nIndexBits] = {};
    for (int i = 0; i < nIndexBits; ++i)
    {
        bitOrdering[i] = i;
    }
    const int bitStringLen = nIndexBits;
    custatevecIndex_t bitStrings[nShots];
    double randnums[nShots] = {};
    generateRandomArray(randnums, nShots);

    cuType xMat[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};
    cuType zMat[] = {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {-1.0, 0.0}};
    cuType hMat[] = {{INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {INV_SQRT2, 0.0}, {-INV_SQRT2, 0.0}};

    cuType *d_sv;
    CHECK_CUDA(hipMallocManaged((void **)&d_sv, svSize * sizeof(cuType)));

    //----------------------------------------------------------------------------------------------

    {
        auto start_m = std::chrono::high_resolution_clock::now();

        int controlsAll[nIndexBits];
        int controlsAllExceptLast[nIndexBits - 1];
        int markTargets[] = {nIndexBits - 1};
        for (int i = 0; i < nIndexBits - 1; ++i)
        {
            controlsAll[i] = i;
            controlsAllExceptLast[i] = i;
        }
        controlsAll[nIndexBits - 1] = nIndexBits - 1;

        // custatevec handle initialization
        custatevecSamplerDescriptor_t sampler;
        custatevecHandle_t handle;
        CHECK_CUSTATEVECTOR(custatevecCreate(&handle));
        void *extraWorkspace = nullptr;
        size_t extraWorkspaceSizeInBytes = 0;

        // Init to zero state
        d_sv[0] = {1, 0};
        for (int i = 1; i < svSize; ++i)
        {
            d_sv[i] = {0, 0};
        }
        // H to all qubits
        for (int i = 0; i < nIndexBits; ++i)
        {
            int targets[] = {i};
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
        }
        // H to all qubits

        for (int i = 0; i < 10; ++i)
        {
            // mark
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, zMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast, nullptr,
                nIndexBits - 1, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            // Diffusion
            // H->all, X->all, cz->allexceptLast mark, x->all, H->all
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, xMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                handle, d_sv, cuStateVecCudaDataType, nIndexBits, zMat, cuStateVecCudaDataType,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, markTargets, 1, controlsAllExceptLast, nullptr,
                nIndexBits - 1, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, xMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
            for (int j = 0; j < nIndexBits; ++j)
            {
                int targets[] = {j};
                CHECK_CUSTATEVECTOR(custatevecApplyMatrix(
                    handle, d_sv, cuStateVecCudaDataType, nIndexBits, hMat, cuStateVecCudaDataType,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, targets, 1, {}, nullptr,
                    0, cuStateVecComputeType, extraWorkspace, extraWorkspaceSizeInBytes));
            }
        }

        // create sampler and check the size of external workspace
        CHECK_CUSTATEVECTOR(custatevecSamplerCreate(
            handle, d_sv, cuStateVecCudaDataType, nIndexBits, &sampler, nMaxShots,
            &extraWorkspaceSizeInBytes));

        // allocate external workspace if necessary
        if (extraWorkspaceSizeInBytes > 0)
            CHECK_CUDA(hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes));

        // sample preprocess
        CHECK_CUSTATEVECTOR(custatevecSamplerPreprocess(
            handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes));

        // std::cout << nShots << "\n";
        // std::cout << bitStringLen << "\n";

        //     for(int k = 0; k < nShots; ++k)
        //     {
        //         // std::cout << randnums[k] << "\n";
        //         // std::cout << bitStrings[k] << "\n";

        //     }
        //     for(int k = 0; k < nIndexBits; ++k)
        //     {
        //         std::cout << bitOrdering[k] << "\n";

        //     }

        // sample bit strings
        CHECK_CUSTATEVECTOR(custatevecSamplerSample(
            handle, sampler, bitStrings, bitOrdering, bitStringLen, randnums, nShots,
            CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER));

        // destroy descriptor and handle
        CHECK_CUSTATEVECTOR(custatevecSamplerDestroy(sampler));

        //  destroy handle
        CHECK_CUSTATEVECTOR(custatevecDestroy(handle));
        if (extraWorkspaceSizeInBytes)
            CHECK_CUDA(hipFree(extraWorkspace));

        auto stop_m = std::chrono::high_resolution_clock::now();
        auto duration_m = std::chrono::duration<double>(stop_m - start_m);
        std::cout <<"Time = " << duration_m.count() << std::endl;
    }
    //----------------------------------------------------------------------------------------------

    // printDeviceArray(d_sv, svSize);
    CHECK_CUDA(hipFree(d_sv));

    return EXIT_SUCCESS;
}